﻿#include "hip/hip_runtime.h"
#include <stdio.h>

const int LZ77windowBits = 11;
const int LZ77matchBits = 16 - LZ77windowBits;
const int LZ77windowMask = (1 << LZ77windowBits) - 1;
const int LZ77matchMask = (1 << LZ77matchBits) - 1;
const int LZ77windowSize = 1 << LZ77windowBits;
const int LZ77matchSize = 1 << LZ77matchBits;

__device__ int equalLength(const int* a, const int* b, const int limit)
{
    int result = 0;

    while (result < limit)
        if (*a++ == *b++)
            result++;
        else
            break;

    return result;
}

__global__ void LZ77match(const int* input, int* offset, int* length, const int size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size)
    {
        const int limit = min(size - i - 1, LZ77matchSize);
        const int maxOffset = min(i, LZ77windowSize);

        offset[i] = 0;
        length[i] = 0;

        for (int o = 1; o <= maxOffset; o++)
        {
            const int l = equalLength(input + i, input + i - o, limit);

            if (length[i] < l)
            {
                length[i] = l;
                offset[i] = o;
            }
        }
    }
}

__global__ void LZ77encode(const int* input, int* offset, int* length, const int inputSize, unsigned char* output, int* outputSize)
{
    *outputSize = 0;

    for (int i = 0; i < inputSize; i++)
    {
        if (length[i] > 2)
        {
            const unsigned short block = ((offset[i] - 1) & LZ77windowMask) | (((length[i] - 1) & LZ77matchMask) << LZ77windowBits);

            i += length[i];

            output[(*outputSize)++] = input[i] & 0x7f | 0x80;
            output[(*outputSize)++] = block;
            output[(*outputSize)++] = block >> 8;
        }
        else
        {
            output[(*outputSize)++] = input[i] & 0x7f;
        }
    }
}

__global__ void LZ77decode(const unsigned char* input, int inputSize, int* output, int* outputSize)
{
    int* start = output;
    const unsigned char* end = input + inputSize;

    while (input < end)
    {
        if (*input & 0x80)
        {
            const unsigned short block = input[1] | (input[2] << 8);

            int* base = output - ((block & LZ77windowMask) + 1);
            int length = ((block >> LZ77windowBits) & LZ77windowMask) + 1;

            while(length-- > 0)
            {
                *output = *base;
                output++;
                base++;
            }

            *output = *input & 0x7f;
            output++;

            input += 3;
        }
        else
        {
            *output = *input;
            output++;

            input++;
        }
    }

    *outputSize = output - start;
}

const int maxInputSize = 1000000;
int intData[maxInputSize];
unsigned char byteData[maxInputSize];
int intSize = 0;
int byteSize = 0;

void readDecodedData(const char* file, int* data, int* size)
{
    FILE* stream = fopen(file, "rt");

    *size = 0;
    while (fscanf(stream, "%i", data + *size) > 0)
        (*size)++;

    fclose(stream);
}

void writeDecodedData(const char* file, const int* data, const int size)
{
    FILE* stream = fopen(file, "wt");

    for (int i = 0; i < size; i++)
        fprintf(stream, "%i\n", data[i]);

    fclose(stream);
}

void readEncodedData(const char* file, unsigned char* data, int* size)
{
    FILE* stream = fopen(file, "rb");

    *size = 0;
    while (!feof(stream))
        *size += fread(data + *size, 1, 1024, stream);

    fclose(stream);
}

void writeEncodedData(const char* file, const unsigned char* data, const int size)
{
    FILE* stream = fopen(file, "wb");

    fwrite(data, 1, size, stream);

    fclose(stream);
}

int main(int argc, const char** argv)
{
    if (argc < 4)
        return -1;

    if (!strcmp(argv[1], "-e"))
    {
        readDecodedData(argv[2], intData, &intSize);

        int* dev_int;
        int* dev_length;
        int* dev_offset;
        unsigned char* dev_byte;
        int* dev_size;

        hipMalloc(&dev_int, intSize * sizeof(int));
        hipMalloc(&dev_length, intSize * sizeof(int));
        hipMalloc(&dev_offset, intSize * sizeof(int));
        hipMalloc(&dev_byte, intSize);
        hipMalloc(&dev_size, sizeof(int));

        hipMemcpy(dev_int, intData, intSize * sizeof(int), hipMemcpyHostToDevice);

        LZ77match<<<(intSize + 511) / 512, 512>>>(dev_int, dev_offset, dev_length, intSize);
        LZ77encode<<<1, 1>>>(dev_int, dev_offset, dev_length, intSize, dev_byte, dev_size);

        hipDeviceSynchronize();

        hipMemcpy(&byteSize, dev_size, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(byteData, dev_byte, byteSize, hipMemcpyDeviceToHost);

        hipFree(dev_int);
        hipFree(dev_length);
        hipFree(dev_offset);
        hipFree(dev_byte);
        hipFree(dev_size);

        hipDeviceReset();

        writeEncodedData(argv[3], byteData, byteSize);
    }
    else if (!strcmp(argv[1], "-d"))
    {
        readEncodedData(argv[2], byteData, &byteSize);

        unsigned char* dev_byte;
        int* dev_int;
        int* dev_size;

        hipMalloc(&dev_byte, byteSize);
        hipMalloc(&dev_int, maxInputSize * sizeof(int));
        hipMalloc(&dev_size, sizeof(int));

        hipMemcpy(dev_byte, byteData, byteSize, hipMemcpyHostToDevice);

        LZ77decode<<<1, 1>>>(dev_byte, byteSize, dev_int, dev_size);

        hipDeviceSynchronize();

        hipMemcpy(&intSize, dev_size, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(intData, dev_int, intSize * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(dev_byte);
        hipFree(dev_int);
        hipFree(dev_size);

        hipDeviceReset();

        writeDecodedData(argv[3], intData, intSize);
    }
    else
        return -1;

    return 0;
}
